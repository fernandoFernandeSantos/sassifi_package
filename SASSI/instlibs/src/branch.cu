#include "hip/hip_runtime.h"
/*********************************************************************************** \
 * Copyright (c) 2015, NVIDIA open source projects
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * - Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * 
 * - Neither the name of SASSI nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * This is a SASSI instrumentation library for gathering branch statistics.  It 
 * corresponds to Case Study I in,
 *
 *   "Flexible Software Profiling of GPU Architectures"
 *   Stephenson et al., ISCA 2015.
 *  
 * The application code the user instruments should be instrumented with the
 * following SASSI flag: -Xptxas --sassi-inst-before="cond-branches" \
 *                       -Xptxas --sassi-before-args="cond-branch-info".
 *
 * In addition, be sure to link your application with flags necessary to 
 * hijack "main" and "exit".  You can trivially do this using GNU tools with
 *
 *       -Xlinker "--wrap=main" -Xlinker "--wrap=exit"
 *
 * This will cause calls to main and exit to be replaced by calls to 
 * __wrap_exit(int status) and __wrap_main(int argc, char **argv), which we have
 * defined below.  This allows us to do initialization and finalization without
 * having to worry about object constructor and destructor orders.
 *
 * This version of the library also lets us correlate SASS location to the
 * corresponding CUDA source locations.  To use this feature, you must 
 * compile your application with the "-lineinfo" option.
 *
 * See the branch example in example/Makfile for all the flags you should use.
 *
\***********************************************************************************/

#define __STDC_FORMAT_MACROS
#include <assert.h>
#include <cupti.h>
#include <inttypes.h>
#include <map>
#include <stdint.h>
#include <stdio.h>
#include <string>
#include <unistd.h>
#include "sassi_intrinsics.h"
#include "sassi_dictionary.hpp"
#include "sassi_srcmap.hpp"
#include <sassi/sassi-core.hpp>
#include <sassi/sassi-branch.hpp>

struct BranchCounter {
  uint64_t address;
  int32_t branchType;                    // The branch type.
  int32_t taggedUnanimous;               // Branch had .U modifier, so compiler knows...
  unsigned long long totalBranches;
  unsigned long long takenThreads;
  unsigned long long takenNotThreads;
  unsigned long long divergentBranches;   // Not all branches go the same way.
  unsigned long long activeThreads;       // Number of active threads.
};                                        

static sassi::src_mapper *sassiMapper;

// The actual dictionary of counters, where the key is a branch's PC, and
// the value is the set of counters associated with it.
static __managed__ sassi::dictionary<uint64_t, BranchCounter> *sassiStats;

// Convert the SASSIBranchType to a string that we can print.  See the
// CUDA binary utilities webpage for more information about these types.
const char *SASSIBranchTypeAsString[] = {
  "BRX", "BRA", "RET", "EXIT", "SYNC", "OTHER"
};


///////////////////////////////////////////////////////////////////////////////////
///
///  Collect the stats and print them out before the device counters are reset.
///
///////////////////////////////////////////////////////////////////////////////////
static void sassi_finalize(__attribute__((unused)) sassi::cupti_wrapper *wrapper, 
			   __attribute__((unused)) const CUpti_CallbackData *cb)
{
  // This function will be called either when 1) the device is reset, or 2) the
  // the program is about to exit.  Let's check to see whether the sassiStats
  // map is still valid.  For instance, the user could have reset the device 
  // before the program exited, which would essentially invalidate all device
  // data. (In fact, explicitly reseting the device before program exit is
  // considered best practice.)
  if (sassiMapper->is_device_state_valid())
  {
    FILE *fRes = fopen("sassi-branch.txt", "w");
    
    fprintf(fRes, "%-16.16s %-10.10s %-10.10s %-10.10s %-10.10s %-10.10s %-8.8s %-8.8s Location\n",
	    "Address", "Total/32", "Dvrge/32", "Active", "Taken", "NTaken", 
	    "Type", ".U");

    // Get the SASS PUPC to source code line mapping.
    auto const locMapper = sassiMapper->get_location_map();
    
    sassiStats->map([fRes,&locMapper](uint64_t& pupc, BranchCounter& val) {
	assert(val.address == pupc);
	
	fprintf(fRes, "%-16.16" PRIx64 
		" %-10.llu %-10.llu %-10.llu %-10.llu %-10.llu %-8.4s %-8.d ",
		pupc,
		val.totalBranches, 
		val.divergentBranches,
		val.activeThreads,
		val.takenThreads,
		val.takenNotThreads,
		SASSIBranchTypeAsString[val.branchType],
		val.taggedUnanimous
		);      

	// See if there is a source code mapping for this PUPC.  If you 
	// compiled your code with "-lineinfo" there should be a valid
	// mapping.
	auto it = locMapper.find(pupc);
	if (it != locMapper.end()) {
	  fprintf(fRes, "%s, line %d\n", it->second.file_name->c_str(), it->second.line_num);
	} else {
	  fprintf(fRes, "\n");
	}
      });
  
    fclose(fRes);
  }
}

///////////////////////////////////////////////////////////////////////////////////
/// 
///  We will compile our application using ld's --wrap option, which in this
///  case lets us replace calls to "exit" with calls to "__wrap_exit".  See
///  the make target "ophist-fermi" in ./example/Makefile to see how this
///  is done.
///
///  This should allow us to perform CUDA operations before the CUDA runtime
///  starts shutting down.  In particular, we want to copy our
///  "dynamic_instr_counts" off the device.  If we used UVM, this would happen
///  automatically for us.  But since we don't have the luxury of using UVM
///  for Fermi, we have to make sure that the CUDA runtime is still up and
///  running before trying to issue a hipMemcpy.  Hence these shenanigans.
/// 
///////////////////////////////////////////////////////////////////////////////////
extern "C" void __real_exit(int status);
extern "C" void __wrap_exit(int status)
{
  sassi_finalize(NULL, NULL);
  __real_exit(status);
}

///////////////////////////////////////////////////////////////////////////////////
/// 
///  For programs that don't call exit explicitly, let's catch the fallthrough.
/// 
///////////////////////////////////////////////////////////////////////////////////
extern "C" int __real_main(int argc, char **argv);
extern "C" int __wrap_main(int argc, char **argv)
{
  // Initialize a src_mapper to give us SASS PC->CUDA line mappings.
  sassiMapper = new sassi::src_mapper();

  // Initialize a hashmap to keep track of statistics of branches.  The key
  // is the PC, the value is a BranchCounter.
  sassiStats = new sassi::dictionary<uint64_t, BranchCounter>();

  // Whenever the device is reset, be sure to print out the counters before
  // they are clobbered.
  sassiMapper->register_callback(sassi::cupti_wrapper::event_type::DEVICE_RESET, 
				 sassi::cupti_wrapper::callback_before,
				 sassi_finalize);

  int ret = __real_main(argc, argv);
  sassi_finalize(NULL, NULL);
  return ret;
}

///////////////////////////////////////////////////////////////////////////////////
//
/// This function will be inserted before every conditional branch instruction.
//
///////////////////////////////////////////////////////////////////////////////////
__device__ void sassi_before_handler(SASSIBeforeParams *bp, SASSICondBranchParams *brp) 
{
  // Find out thread index within the warp.
  int threadIdxInWarp = get_laneid();

  // Get masks and counts of 1) active threads in this warp,
  // 2) threads that take the branch, and
  // 3) threads that do not take the branch.
  int active = __ballot(1);
  bool dir = brp->GetDirection();
  int taken = __ballot(dir == true);
  int ntaken = __ballot(dir == false);
  int numActive = __popc(active);
  int numTaken = __popc(taken);
  int numNotTaken = __popc(ntaken);
  bool divergent = (numTaken != numActive && numNotTaken != numActive);

  // The first active thread in each warp gets to write results.
  if ((__ffs(active)-1) == threadIdxInWarp) {
    // Get the address, we'll use it for hashing.
    uint64_t instAddr = bp->GetPUPC();
    
    // Looks up the counters associated with 'instAddr', but if no such entry
    // exits, initialize the counters in the lambda.
    BranchCounter *stats = (*sassiStats).getOrInit(instAddr, [instAddr,brp](BranchCounter* v) {
	v->address = instAddr;
	v->branchType = brp->GetType();
	v->taggedUnanimous = brp->IsUnanimous();
      });

    // Why not sanity check the hash map?
    assert(stats->address == instAddr);
    assert(numTaken + numNotTaken == numActive);

    // Increment the various counters that are associated
    // with this instruction appropriately.
    atomicAdd(&(stats->totalBranches), 1ULL);
    atomicAdd(&(stats->activeThreads), numActive);
    atomicAdd(&(stats->takenThreads), numTaken);
    atomicAdd(&(stats->takenNotThreads), numNotTaken);
    atomicAdd(&(stats->divergentBranches), divergent);
  }
}



