#include "hip/hip_runtime.h"
/*********************************************************************************** \
 * Copyright (c) 2015, NVIDIA open source projects
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * - Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * 
 * - Neither the name of SASSI nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * This example computes, for each instruction, for each destination operand,
 * for each bit in the operand, whether or not the bit is constant over all
 * threads and throughout the course of execution of the program.  It also
 * tracks whether an operand is scalar.  
 *
 * The example is based on case study III in the paper,
 *
 *   "Flexible Software Profiling of GPU Architectures"
 *   Stephenson et al., ISCA 2015.
 *
 * The application code the user instruments should be instrumented with the
 * following SASSI flag: -Xptxas --sassi-inst-after="reg-writes"
 *                       -Xptxas --sassi-after-args="reg-info"
 *            [optional] -Xptxas --sassi-iff-true-predicate-handler-call
 *  
\***********************************************************************************/

#define __STDC_FORMAT_MACROS
#include <algorithm>
#include <assert.h>
#include <cupti.h>
#include <inttypes.h>
#include <list>
#include <map>
#include <stdint.h>
#include <stdio.h>
#include <string>
#include <unistd.h>
#include "sassi_intrinsics.h"
#include "sassi_dictionary.hpp"
#include "sassi_lazyallocator.hpp"
#include <sassi/sassi-core.hpp>
#include <sassi/sassi-regs.hpp>


///////////////////////////////////////////////////////////////////////////////////
///  
///  Each SASS operation has a number of DSTOperands.  This class
///  keeps track of the stats for a given operand.
///
///////////////////////////////////////////////////////////////////////////////////
class DSTOperand {
public:
  ///////////////////////////////////////////////////////////////////
  ///  
  ///  Initialize the counters for this operand.
  ///
  ///////////////////////////////////////////////////////////////////
  static __device__ void init(DSTOperand *op)
  {
    op->isScalar = -1;
    op->constantOnes = -1;
    op->constantZeros = -1;
  }

  ///////////////////////////////////////////////////////////////////
  ///  
  ///  Prints the bitvector to the given file.
  ///
  ///////////////////////////////////////////////////////////////////
  void print_bits(FILE *f) const
  {
    for (int bn = 31; bn >= 0; bn--) {
      int oneBit = ((constantOnes >> bn) & 0x1);
      int zeroBit = ((constantZeros >> bn) & 0x1);
      if (oneBit == 0 && zeroBit == 0) {
	fprintf(f, "T");
      }
      else if (oneBit == 0 && zeroBit == 1) {
	fprintf(f, "0"); 
      }
      else if (oneBit == 1 && zeroBit == 0) {
	fprintf(f, "1"); 
      }
      else if (oneBit == 1 && zeroBit == 1) {
	fprintf(f, "X");
      }
    }
  }

  ///////////////////////////////////////////////////////////////////
  ///  
  ///  Prints the operand stats to the given file.
  ///
  ///////////////////////////////////////////////////////////////////
  void print(FILE *f)
  {
    fprintf(f, "[%d, \"%s\", %s, [", 
	    regNum, 
	    SASSITypeAsString[regType],
	    isScalar ? "SCALAR " : "VARIANT");
    print_bits(f);
    fprintf(f, "]]");
  }
  
  int regNum;
  int isScalar;
  SASSIType regType;
  int constantOnes;
  int constantZeros;
};


///////////////////////////////////////////////////////////////////////////////////
///  
///  Keep the statistics for each SASS operation.
///
///////////////////////////////////////////////////////////////////////////////////
class SASSOp {
public:
  // This is a bit dirty, and deserves some explanation.  Until
  // device-side allocation of memory matures, we are going to
  // pre-allocate space for the SASSOp statistics on the host.
  // In so doing, we will need to account for the worst-case
  // SASS instruction with regard to number of destination 
  // operands. 
#define MAX_DST_OPERANDS 4

  ///////////////////////////////////////////////////////////////////
  ///  
  ///  Initialize the SASSOp passed in.
  ///
  ///////////////////////////////////////////////////////////////////
  __device__ static void init(SASSOp *op, SASSIRegisterParams *rp)
  {
    op->weight = 0;
    op->numDsts = rp->GetNumGPRDsts();
    assert(op->numDsts <= MAX_DST_OPERANDS);

    // Initialize all of the fields appropriately.
    for (int i = 0; i < op->numDsts; i++) {
      DSTOperand::init(&(op->operands[i]));
      SASSIRegisterParams::GPRRegInfo regInfo = rp->GetGPRDst(i);
      op->operands[i].regNum = rp->GetRegNum(regInfo);
      op->operands[i].regType = rp->GetRegType(regInfo);
    }
  }
  
  ///////////////////////////////////////////////////////////////////
  ///  
  ///  Prints the operation stats to the given file.
  ///
  ///////////////////////////////////////////////////////////////////
  void print(FILE *f)
  {
    fprintf(f, "%lld, [", weight);
    for (int i = 0; i < numDsts; i++) {
      operands[i].print(f);
    }
    fprintf(f, "]");
  }
  
  unsigned long long weight;
  int numDsts;
  DSTOperand operands[MAX_DST_OPERANDS];
};


/// The actual dictionary, declared as a UVM managed type so we can access it on 
/// the device and host.
static __managed__ sassi::dictionary<uint64_t, SASSOp> *sassi_stats;


///////////////////////////////////////////////////////////////////////////////////
///  
///  We will register this function to be called whenever the device is reset, 
///  or when the program is about to exit.  The function will print out the 
///  aggregated statistics.
///
///////////////////////////////////////////////////////////////////////////////////
static void sassi_finalize(sassi::lazy_allocator::device_reset_reason reason)
{
  struct KVTuple {
    uint64_t k;
    SASSOp *v;
  };

  FILE *resultsFile = fopen("sassi-valueprof.txt", "w");
  
  fprintf(resultsFile, "\nValue profiling results\n");
  fprintf(resultsFile, "ADDRESS | WEIGHT | [regnum, type, scalarness, bitstring]*\n");
  fprintf(resultsFile, "---------------------------------------------------------\n");
  
  std::vector<KVTuple> ops;
  sassi_stats->map([&ops](uint64_t& key, SASSOp &val) {
      ops.push_back({key, &val});
    });
  
  std::sort(ops.begin(), ops.end(), [](KVTuple a, const KVTuple b) {
      return a.k < b.k;
    });
  
  for (KVTuple t : ops) {
    fprintf(resultsFile, "[%.16" PRIx64 ", ", t.k);
    t.v->print(resultsFile);
    fprintf(resultsFile, "]\n");
  }
  
  hipDeviceSynchronize();
  fclose(resultsFile);
}

///////////////////////////////////////////////////////////////////////////////////
///
///  Lazily allocate a dictionary before the first kernel launch.
///
///////////////////////////////////////////////////////////////////////////////////
static sassi::lazy_allocator mapAllocator([]() {
    sassi_stats = new sassi::dictionary<uint64_t, SASSOp>();
  }, sassi_finalize);


///////////////////////////////////////////////////////////////////////////////////
//
//  This example uses the atomic bitwise operations to keep track of the constant
//  bits produced by each instruction.
//
///////////////////////////////////////////////////////////////////////////////////
__device__ void sassi_after_handler(SASSIAfterParams* ap, SASSIRegisterParams *rp)
{
  int threadIdxInWarp = get_laneid();
  int firstActiveThread = (__ffs(__ballot(1))-1); /*leader*/

  // Get the "probably unique" PC.
  uint64_t pupc = ap->GetPUPC();

  // The dictionary will return the SASSOp associated with this PC, or insert
  // it if it does not exist.  If it does not exist, the lambda passed as
  // the second argument to getOrInit is used to initialize the SASSOp.
  SASSOp *stats = sassi_stats->getOrInit(pupc, [&rp](SASSOp *v) {
      SASSOp::init(v, rp);
    });
  
  // Record the number of times the instruction executes.
  atomicAdd(&(stats->weight), 1);
  for (int d = 0; d < rp->GetNumGPRDsts(); d++) {
    // Get the value in each destination register.
    SASSIRegisterParams::GPRRegInfo regInfo = rp->GetGPRDst(d);
    SASSIRegisterParams::GPRRegValue regVal = rp->GetRegValue(ap, regInfo); 

    // Use atomic AND operations to track constant bits.
    atomicAnd(&(stats->operands[d].constantOnes), regVal.asInt); 
    atomicAnd(&(stats->operands[d].constantZeros), ~regVal.asInt);

    int leaderValue = __shfl(regVal.asInt, firstActiveThread); 
    int allSame = (__all(regVal.asInt == leaderValue) != 0);
    // The warp leader gets to write results.
    if (threadIdxInWarp == firstActiveThread) { 
      atomicAnd(&(stats->operands[d].isScalar), allSame);
    }
  }
}




